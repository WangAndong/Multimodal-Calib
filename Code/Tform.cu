#include "Tform.h"

Camera::Camera(bool panoramic):
	panoramic_(panoramic){
	size_t camMemSize = CAM_WIDTH * CAM_HEIGHT * sizeof(float);
	CudaSafeCall(hipMalloc((void**)&d_camera_, camMemSize));
	CudaSafeCall(hipMemset(d_camera_, 0, camMemSize));
}

Camera::~Camera(void){
	 CudaSafeCall(hipFree(d_camera_));
}

void Camera::SetCam(float* cam){
	size_t tformMemSize = CAM_WIDTH * CAM_HEIGHT * sizeof(float);
	 CudaSafeCall(hipMemcpy(d_camera_, cam, tformMemSize, hipMemcpyHostToDevice));
}

float* Camera::d_GetCam(void){
	return d_camera_;
}

bool Camera::IsPanoramic(void){
	return panoramic_;
}


Tform::Tform(size_t sizeTform):
	sizeTform_(sizeTform)
{
	size_t tformMemSize = sizeTform * sizeTform * sizeof(float);
	 CudaSafeCall(hipMalloc((void**)&d_tform_, tformMemSize));
	 CudaSafeCall(hipMemset(d_tform_, 0, tformMemSize));
}

Tform::~Tform(void){
	 CudaSafeCall(hipFree(d_tform_));
}

void Tform::SetTform(float* tform){
	size_t tformMemSize = sizeTform_ * sizeTform_ * sizeof(float);

	 CudaSafeCall(hipMemcpy(d_tform_, tform, tformMemSize, hipMemcpyHostToDevice));
}

float* Tform::d_GetTform(void){
	return d_tform_;
}

CameraTform::CameraTform(Camera* cam):
	Tform(CAM_DIM + 1){
	cam_ = cam;
}


void CameraTform::d_Transform(SparseScan* in, SparseScan* out){

	if(out->getNumPoints() < in->getNumPoints()){
		TRACE_ERROR("output is too small to hold inputs points, returning");
		return;
	}
	if(in->getNumDim() != CAM_DIM){
		TRACE_ERROR("camera transform can only operate on a 3d input, returning untransformed points");
		CudaSafeCall(hipMemcpy(out->GetLocation()->GetGpuPointer(), in->GetLocation()->GetGpuPointer(), in->getNumDim()*in->getNumPoints()*sizeof(float), hipMemcpyDeviceToDevice));
		return;
	}
	if(cam_ == NULL){
		TRACE_ERROR("camera transform requires a setup camera, returning untransformed points");
		 CudaSafeCall(hipMemcpy(out->GetLocation()->GetGpuPointer(), in->GetLocation()->GetGpuPointer(), in->getNumDim()*in->getNumPoints()*sizeof(float), hipMemcpyDeviceToDevice));
		return;
	}

	CameraTransformKernel<<<gridSize(in->getDimSize(0)), BLOCK_SIZE>>>
		(d_tform_, cam_->d_GetCam(), (float*)in->GetLocation()->GetGpuPointer(), (float*)out->GetLocation()->GetGpuPointer(), in->getDimSize(0), cam_->IsPanoramic());
	CudaCheckError();
}

AffineTform::AffineTform(void):
	Tform(AFFINE_DIM + 1){}

void AffineTform::d_Transform(SparseScan* in, SparseScan* out){

	if(out->getNumPoints() < in->getNumPoints()){
		TRACE_ERROR("output is too small to hold inputs points, returning");
		return;
	}
	if(in->getNumDim() != AFFINE_DIM){
		TRACE_ERROR("affine transform can only operate on a 2d input, returning untransformed points");
		CudaSafeCall(hipMemcpy(out->GetLocation()->GetGpuPointer(), in->GetLocation()->GetGpuPointer(), in->getNumDim()*in->getNumPoints()*sizeof(float), hipMemcpyDeviceToDevice));
		return;
	}

	AffineTransformKernel<<<gridSize(in->getDimSize(0)), BLOCK_SIZE>>>(d_tform_, (float*)in->GetLocation()->GetGpuPointer(), (float*)out->GetLocation()->GetGpuPointer(), in->getDimSize(0));
	 //CudaSafeCall(hipMemcpy(out->GetLocation()->GetGpuPointer(), in->GetLocation()->GetGpuPointer(), in->getNumPoints()*sizeof(float), hipMemcpyDeviceToDevice));
}