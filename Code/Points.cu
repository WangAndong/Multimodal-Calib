#include "Points.h"
#include "Kernel.h"

#include "CI\code\cubicPrefilter3D.cu"
#include "CI\code\cubicTex3D.cu"

float* PointsList::PointsSetup(float* points, const size_t numEntries, bool copy){
	if(copy){
		float* out = new float[numEntries];
		for(size_t i = 0; i < numEntries; i++){
			out[i] = points[i];
		}
		return out;
	}
	else{
		return points;
	}
}

PointsList::PointsList(float* points, const size_t numEntries, bool copy):
	points_(PointsSetup(points,numEntries,copy)),
	d_points_(NULL),
	numEntries_(numEntries){
		TRACE_INFO("%i points set",numEntries);  
}

PointsList::PointsList(const size_t numEntries):
	numEntries_(numEntries),
	d_points_(NULL),
	points_(new float[numEntries]){
		TRACE_INFO("%i points set",numEntries);
	}

PointsList::~PointsList(){
	if(IsOnGpu()){
		ClearGpu();
	}

	delete [] points_;
}

size_t PointsList::GetNumEntries(){
	return numEntries_;
}

float* PointsList::GetCpuPointer(){
	return points_;
}

void* PointsList::GetGpuPointer(){
	if(!IsOnGpu()){
		TRACE_WARNING("points were not on GPU, creating gpu pointer first");
		AllocateGpu();
	}
	return d_points_;
}

bool PointsList::IsOnGpu(){
	return (d_points_ != NULL);
}
	
void PointsList::AllocateGpu(void){
	if(d_points_ != NULL){
		TRACE_WARNING("d_points_ already full, clearing and overwriting");
		ClearGpu();
	}
	CudaSafeCall(hipMalloc((void**)&(d_points_), sizeof(float)*numEntries_));
}

void PointsList::ClearGpu(void){
	if(IsOnGpu()){
		CudaSafeCall(hipFree(d_points_));
		d_points_ = NULL;
	}
	else{
		TRACE_WARNING("nothing on gpu to clear");
	}
}

void PointsList::GpuToCpu(void){
	if(IsOnGpu()){
		CudaSafeCall(hipMemcpy(points_, d_points_, numEntries_*sizeof(float), hipMemcpyDeviceToHost));
	}
	else {
		TRACE_ERROR("No memory was allocated on gpu, returning");
	}
}

void PointsList::CpuToGpu(void){
	if(!IsOnGpu()){
		TRACE_WARNING("No memory was allocated on gpu, allocating now");
		AllocateGpu();
	}
	TRACE_INFO("%i points to be copied from host to device", numEntries_);
	CudaSafeCall(hipMemcpy(d_points_, points_, sizeof(float)*numEntries_, hipMemcpyHostToDevice));
}

void TextureList::AllocateGpu(void){
	if(d_points_ != NULL){
		TRACE_WARNING("d_points_ already full, clearing and overwriting");
		ClearGpu();
	}
	const hipExtent extent = make_hipExtent(sizeof(float)*width_, height_, depth_);
	d_points_ = new hipPitchedPtr;
	CudaSafeCall(hipMalloc3D(((hipPitchedPtr*)d_points_), extent));
}

void TextureList::GpuToCpu(void){
	if(IsOnGpu()){
		const hipExtent extent = make_hipExtent(sizeof(float)*width_, height_, depth_);
		hipPitchedPtr points = make_hipPitchedPtr( (void*)points_, sizeof(float), width_*height_*depth_, 1);

		hipMemcpy3DParms copyParams = {0};
		copyParams.dstPtr = points;
		copyParams.extent = extent;
		copyParams.kind = hipMemcpyDeviceToHost;

		if(texInMem_){
			copyParams.srcArray = (hipArray_t)d_points_;
		}
		else {
			copyParams.srcPtr = *((hipPitchedPtr*)d_points_);
		}

		CudaSafeCall( hipMemcpy3D(&copyParams) ); 
	}
	else {
		TRACE_ERROR("No memory was allocated on gpu, returning");
	}
}

void TextureList::CpuToGpu(void){
	if(!IsOnGpu()){
		TRACE_WARNING("No memory was allocated on gpu, allocating now");
		AllocateGpu();
	}
	TRACE_INFO("%i points to be copied from host to device", numEntries_);
	const hipExtent extent = make_hipExtent(sizeof(float)*width_, height_, depth_);
	
	hipPitchedPtr points = make_hipPitchedPtr( (void*)points_, sizeof(float), width_*height_*depth_, 1);

	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr = points;
	copyParams.extent = extent;
	copyParams.kind = hipMemcpyHostToDevice;

	if(texInMem_){
		copyParams.dstArray = (hipArray_t)d_points_;
	}
	else {
		copyParams.dstPtr = *((hipPitchedPtr*)d_points_);
	}

	CudaSafeCall( hipMemcpy3D(&copyParams) ); 
}

void TextureList::ClearGpu(void){
	if(IsOnGpu()){
		if(texInMem_){
			CudaSafeCall(hipFreeArray((hipArray_t)d_points_));
		}
		else {
			CudaSafeCall(((hipPitchedPtr*)d_points_)->ptr);
			free(d_points_);
		}
		d_points_ = NULL;
		texInMem_ = false;
	}
	else{
		TRACE_WARNING("nothing on gpu to clear");
	}
}

TextureList::TextureList(float* points, bool copy, const size_t height, const size_t width, const size_t depth):
	PointsList(points, height*width*depth, copy),
	height_(height),
	width_(width),
	depth_(depth),
	texInMem_(false){

	AllocateGpu();
	CpuToGpu();
	PrefilterArray();
	ArrayToTexture();
}

size_t TextureList::GetHeight(void){
	return height_;
}

size_t TextureList::GetWidth(void){
	return width_;
}

size_t TextureList::GetDepth(void){
	return depth_;
}

void TextureList::ArrayToTexture(void){

	const hipExtent extent = make_hipExtent(width_, height_, depth_);

	// Create the B-spline coefficients texture
	hipChannelFormatDesc channelDescCoeff = hipCreateChannelDesc<float>();
	
	hipArray* temp;
	
	CudaSafeCall(hipMalloc3DArray(&temp, &channelDescCoeff, extent));
	//CudaSafeCall(hipMemcpyToArray(temp, 0, 0, d_points_, (sizeof(float) * width_ * height_ * depth_), hipMemcpyDeviceToDevice));
	//CudaSafeCall(hipBindTextureToArray(tex, temp, *channelDescCoeff));
	
	texInMem_ = true;
	tex.normalized = false;  // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;

	//stores texture
	hipFree(d_points_);
	d_points_ = temp;
}

void TextureList::PrefilterArray(void){
	if(!IsOnGpu()){
		TRACE_WARNING("Gpu must be set up for filtering, allocating memory and copying data now");
		AllocateGpu();
		CpuToGpu();
	}
		
	//inialize texture values
	//this may have red underlines everywhere but it is right
	CubicBSplinePrefilter3D((float*)d_points_, sizeof(float), width_, height_, depth_);

	GpuToCpu();
}