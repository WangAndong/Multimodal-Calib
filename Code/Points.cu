#include "Points.h"
#include "Kernel.h"

float* PointsList::PointsSetup(float* points, const size_t numEntries, bool copy){
	if(copy){
		float* out = new float[numEntries];
		for(size_t i = 0; i < numEntries; i++){
			out[i] = points[i];
		}
		return out;
	}
	else{
		return points;
	}
}

PointsList::PointsList(float* points, const size_t numEntries, bool copy):
	points_(PointsSetup(points,numEntries,copy)),
	d_points_(NULL),
	numEntries_(numEntries){
		TRACE_INFO("%i points set",numEntries);  
}

PointsList::PointsList(const size_t numEntries):
	numEntries_(numEntries),
	d_points_(NULL),
	points_(new float[numEntries]){
		TRACE_INFO("%i points set",numEntries);
	}

PointsList::~PointsList(){
	if(IsOnGpu()){
		ClearGpu();
	}

	delete[] points_;
	points_ = NULL;
}

size_t PointsList::GetNumEntries(){
	return numEntries_;
}

float* PointsList::GetCpuPointer(){
	return points_;
}

void* PointsList::GetGpuPointer(){
	if(!IsOnGpu()){
		TRACE_WARNING("points were not on GPU, creating gpu pointer first");
		AllocateGpu();
	}
	return d_points_;
}

bool PointsList::IsOnGpu(){
	return (d_points_ != NULL);
}
	
void PointsList::AllocateGpu(void){
	if(d_points_ != NULL){
		TRACE_WARNING("d_points_ already full, clearing and overwriting");
		ClearGpu();
	}
	CudaSafeCall(hipMalloc((void**)&(d_points_), sizeof(float)*numEntries_));
}

void PointsList::ClearGpu(void){
	if(IsOnGpu()){
		CudaSafeCall(hipFree(d_points_));
		d_points_ = NULL;
	}
	else{
		TRACE_WARNING("nothing on gpu to clear");
	}
}

void PointsList::GpuToCpu(void){
	if(IsOnGpu()){
		CudaSafeCall(hipMemcpy(points_, d_points_, numEntries_*sizeof(float), hipMemcpyDeviceToHost));
	}
	else {
		TRACE_ERROR("No memory was allocated on gpu, returning");
	}
}

void PointsList::CpuToGpu(void){
	if(!IsOnGpu()){
		TRACE_WARNING("No memory was allocated on gpu, allocating now");
		AllocateGpu();
	}
	TRACE_INFO("%i points to be copied from host to device", numEntries_);
	CudaSafeCall(hipMemcpy(d_points_, points_, sizeof(float)*numEntries_, hipMemcpyHostToDevice));
}

TextureList::~TextureList(){
	if(IsOnGpu()){
		ClearGpu();
	}
}

void TextureList::AllocateGpu(void){
	if(d_points_ != NULL){
		TRACE_WARNING("d_points_ already full, clearing and overwriting");
		ClearGpu();
	}
	const hipExtent extent = make_hipExtent(sizeof(float)*width_, height_, 1);
	d_points_ = new hipPitchedPtr[depth_];
	for(size_t i = 0; i < depth_; i++){
		CudaSafeCall(hipMalloc3D(&(((hipPitchedPtr*)d_points_)[i]), extent));
	}

}

void TextureList::GpuToCpu(void){
	if(IsOnGpu()){

		hipMemcpy3DParms copyParams = {0};
		copyParams.kind = hipMemcpyDeviceToHost;

		if(texInMem_){
			copyParams.extent = make_hipExtent(width_, height_, 1);
		}
		else {
			copyParams.extent = make_hipExtent(sizeof(float)*width_, height_, 1);
			
		}

		for(size_t i = 0; i < depth_; i++){
			copyParams.dstPtr = make_hipPitchedPtr( &(points_[i*width_*height_]), sizeof(float)*width_, width_, height_);

			if(texInMem_){
				copyParams.srcArray = *(((hipArray***)d_points_)[0]);
			}
			else {
				copyParams.srcPtr = ((hipPitchedPtr*)d_points_)[i];
			}
			CudaSafeCall( hipMemcpy3D(&copyParams) );
		}
	}
	else {
		TRACE_ERROR("No memory was allocated on gpu, returning");
	}
}

void TextureList::CpuToGpu(void){
	if(!IsOnGpu()){
		TRACE_WARNING("No memory was allocated on gpu, allocating now");
		AllocateGpu();
	}
	TRACE_INFO("%i points to be copied from host to device", numEntries_);

	hipMemcpy3DParms copyParams = {0};
	copyParams.kind = hipMemcpyHostToDevice;

	if(texInMem_){
		copyParams.extent = make_hipExtent(width_, height_, 1);
	}
	else {
		copyParams.extent = make_hipExtent(sizeof(float)*width_, height_, 1);
			
	}

	for(size_t i = 0; i < depth_; i++){
		copyParams.srcPtr = make_hipPitchedPtr( &(points_[i*width_*height_]), sizeof(float)*width_, width_, height_);	

		if(texInMem_){
			copyParams.dstArray = *((hipArray***)d_points_)[i];
		}
		else {
			copyParams.dstPtr = ((hipPitchedPtr*)d_points_)[i];
		}
		CudaSafeCall( hipMemcpy3D(&copyParams) );
	}
}

void TextureList::ClearGpu(void){
	if(IsOnGpu()){
		for(size_t i = 0; i < depth_; i++){
			if(texInMem_){
				CudaSafeCall(hipFreeArray(*(((hipArray***)d_points_)[i])));
			}
			else {
				CudaSafeCall(hipFree((((hipPitchedPtr*)d_points_)[i]).ptr));
			}
		}

		delete[] d_points_;
		d_points_ = NULL;
		texInMem_ = false;
	}
	else{
		TRACE_WARNING("nothing on gpu to clear");
	}
}

TextureList::TextureList(float* points, bool copy, const size_t width, const size_t height, const size_t depth):
	PointsList(points, height*width*depth, copy),
	height_(height),
	width_(width),
	depth_(depth),
	texInMem_(false){
	AllocateGpu();
	CpuToGpu();
	//PrefilterArray();
	//ArrayToTexture();
}

size_t TextureList::GetHeight(void){
	return height_;
}

size_t TextureList::GetWidth(void){
	return width_;
}

size_t TextureList::GetDepth(void){
	return depth_;
}

void TextureList::ArrayToTexture(void){

	const hipExtent extent = make_hipExtent(width_, height_, depth_);

	// Create the B-spline coefficients texture
	hipChannelFormatDesc channelDescCoeff = hipCreateChannelDesc<float>();
	
	hipArray*** temp = new hipArray**[depth_];
	for(size_t i = 0; i < depth_; i++){
		temp[i] = new hipArray*;
	}

	
	for(size_t i = 0; i < depth_; i++){
		CudaSafeCall(hipMallocArray(temp[i], &channelDescCoeff, width_, height_));
		CudaSafeCall(hipMemcpy2DToArray(*(temp[i]), 0, 0, (((hipPitchedPtr*)d_points_)[i].ptr), (((hipPitchedPtr*)d_points_)[i].pitch), width_ * sizeof(float), height_, hipMemcpyDeviceToDevice));
	}

	tex.normalized = false;  // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;

	//stores texture
	ClearGpu();
	CudaCheckError();
	texInMem_ = true;
	d_points_ = temp;
}

void TextureList::PrefilterArray(void){
	if(!IsOnGpu()){
		TRACE_WARNING("Gpu must be set up for filtering, allocating memory and copying data now");
		AllocateGpu();
		CpuToGpu();
	}
		
	//inialize texture values
	//this may have red underlines everywhere but it is right
	for(size_t i = 0; i < depth_; i++){
		float* ptr = (float*)((((hipPitchedPtr*)d_points_)[i]).ptr);
		//RunBSplineKernel(ptr, width_,height_);
	}

	GpuToCpu();
}