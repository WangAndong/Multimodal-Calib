#include "Points.h"
#include "Kernel.h"

float* PointsList::PointsSetup(float* points, const size_t numEntries, bool copy){
	if(copy){
		float* out = new float[numEntries];
		for(size_t i = 0; i < numEntries; i++){
			out[i] = points[i];
		}
		return out;
	}
	else{
		return points;
	}
}

PointsList::PointsList(float* points, const size_t numEntries, bool copy):
	points_(PointsSetup(points,numEntries,copy)),
	d_points_(NULL),
	numEntries_(numEntries){
		TRACE_INFO("%i points set",numEntries);  
}

PointsList::PointsList(const size_t numEntries):
	numEntries_(numEntries),
	d_points_(NULL),
	points_(new float[numEntries]){
		TRACE_INFO("%i points set",numEntries);
}

PointsList::~PointsList(){
	if(IsOnGpu()){
		ClearGpu();
	}

	delete[] points_;
	points_ = NULL;
}

size_t PointsList::GetNumEntries(){
	return numEntries_;
}

float* PointsList::GetCpuPointer(){
	return points_;
}

void* PointsList::GetGpuPointer(){
	if(!IsOnGpu()){
		TRACE_WARNING("points were not on GPU, creating gpu pointer first");
		AllocateGpu();
	}
	return d_points_;
}

bool PointsList::IsOnGpu(){
	return (d_points_ != NULL);
}
	
void PointsList::AllocateGpu(void){
	if(d_points_ != NULL){
		TRACE_WARNING("d_points_ already full, clearing and overwriting");
		ClearGpu();
	}
	CudaSafeCall(hipMalloc((void**)&(d_points_), sizeof(float)*numEntries_));
}

void PointsList::ClearGpu(void){
	if(IsOnGpu()){
		CudaSafeCall(hipFree(d_points_));
		d_points_ = NULL;
	}
	else{
		TRACE_WARNING("nothing on gpu to clear");
	}
}

void PointsList::GpuToCpu(void){
	if(IsOnGpu()){
		CudaSafeCall(hipMemcpy(points_, d_points_, numEntries_*sizeof(float), hipMemcpyDeviceToHost));
	}
	else {
		TRACE_ERROR("No memory was allocated on gpu, returning");
	}
}

void PointsList::CpuToGpu(void){
	if(!IsOnGpu()){
		TRACE_WARNING("No memory was allocated on gpu, allocating now");
		AllocateGpu();
	}
	TRACE_INFO("%i points to be copied from host to device", numEntries_);
	CudaSafeCall(hipMemcpy(d_points_, points_, sizeof(float)*numEntries_, hipMemcpyHostToDevice));
}

TextureList::~TextureList(){
	if(IsOnGpu()){
		ClearGpu();
	}
}

void TextureList::AllocateGpu(void){
	if(d_points_ != NULL){
		TRACE_WARNING("d_points_ already full, clearing and overwriting");
		ClearGpu();
	}
	CudaSafeCall(hipMalloc((void**)&(d_points_), sizeof(float)*width_*height_*depth_));
}

void TextureList::GpuToCpu(void){
	if(IsOnGpu()){

		if(texInMem_){
			for(size_t i = 0; i < depth_; i++){
				CudaSafeCall(hipMemcpyFromArray(&(((float*)points_)[width_*height_*i]),((hipArray**)d_points_)[i],0,0,width_*height_*sizeof(float), hipMemcpyDeviceToHost));
			}
		}
		else {
			CudaSafeCall(hipMemcpy(points_,((hipArray**)d_points_),sizeof(float)*width_*height_*depth_,hipMemcpyDeviceToHost));
		}
	}
	else {
		TRACE_ERROR("No memory was allocated on gpu, returning");
	}
}

void TextureList::CpuToGpu(void){
	if(!IsOnGpu()){
		TRACE_WARNING("No memory was allocated on gpu, allocating now");
		AllocateGpu();
	}
	TRACE_INFO("%i points to be copied from host to device", numEntries_);

	hipMemcpy3DParms copyParams = {0};
	copyParams.kind = hipMemcpyHostToDevice;

	if(texInMem_){
		for(size_t i = 0; i < depth_; i++){
			CudaSafeCall(hipMemcpyToArray(((hipArray**)d_points_)[i], 0, 0, &(((float*)points_)[width_*height_*i]), width_*height_*sizeof(float), hipMemcpyHostToDevice));
		}
	}
	else {
		CudaSafeCall(hipMemcpy(d_points_,points_,sizeof(float)*width_*height_*depth_,hipMemcpyHostToDevice));
	}
}

void TextureList::ClearGpu(void){
	if(IsOnGpu()){
		
		if(texInMem_){
			for(size_t i = 0; i < depth_; i++){
				hipFreeArray(((hipArray**)d_points_)[i]);
			}
		}
		else {
			CudaSafeCall(hipFree(d_points_));
		}
		d_points_ = NULL;
		texInMem_ = false;
	}
	else{
		TRACE_WARNING("nothing on gpu to clear");
	}
}

TextureList::TextureList(float* points, bool copy, const size_t width, const size_t height, const size_t depth):
	PointsList(points, height*width*depth, copy),
	height_(height),
	width_(width),
	depth_(depth),
	texInMem_(false){
	AllocateGpu();
	CpuToGpu();
	//PrefilterArray();
	ArrayToTexture();
}

size_t TextureList::GetHeight(void){
	return height_;
}

size_t TextureList::GetWidth(void){
	return width_;
}

size_t TextureList::GetDepth(void){
	return depth_;
}

void TextureList::ArrayToTexture(void){

	// Create the B-spline coefficients texture
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
	hipArray** temp = new hipArray*[depth_];
	
	for(size_t i = 0; i < depth_; i++){
		CudaSafeCall(hipMallocArray(&temp[i], &channelDesc, width_, height_));
		CudaSafeCall(hipMemcpyToArray(temp[i], 0, 0, &(((float*)d_points_)[width_*height_*i]), width_*height_*sizeof(float), hipMemcpyDeviceToDevice));
	}

	//stores texture
	ClearGpu();
	CudaCheckError();
	texInMem_ = true;
	d_points_ = temp;
}

void TextureList::PrefilterArray(void){
	if(!IsOnGpu()){
		TRACE_WARNING("Gpu must be set up for filtering, allocating memory and copying data now");
		AllocateGpu();
		CpuToGpu();
	}
		
	//inialize texture values
	//this may have red underlines everywhere but it is right
	//for(size_t i = 0; i < depth_; i++){
	//	float* ptr = (float*)((((hipPitchedPtr*)d_points_)[i]).ptr);
		//RunBSplineKernel(ptr, width_,height_);
	//}


	GpuToCpu();
}