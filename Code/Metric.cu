#include "hip/hip_runtime.h"
#include "Metric.h"
#include "mi.h"
#include "Kernel.h"
#include "reduction.h"

extern "C" float cudaMIa(float* src1, float* src2, int length, int xbins, int ybins, struct cudaHistOptions* p_options, int device, int incZeros);

void Metric::EvalMetric(SparseScan* A, SparseScan* B, float* value){
	*value = 0;
}

MI::MI(size_t bins):
	bins_(bins){
}

void MI::EvalMetric(SparseScan* A, SparseScan* B, float* value){
	
	//check scans exist
	if(A == NULL || B == NULL){
		TRACE_ERROR("Two scans are required for the metric to operate");
		*value = 0;
		return;
	}

	size_t numElements;
	//check scans of same size
	if(A->getPoints()->GetNumEntries() != B->getPoints()->GetNumEntries()){
		numElements = (A->getPoints()->GetNumEntries() > B->getPoints()->GetNumEntries()) ? B->getPoints()->GetNumEntries() : A->getPoints()->GetNumEntries();
		TRACE_WARNING("Number of entries does not match, Scan A has %i, Scan B has %i, only using %i entries",A->getPoints()->GetNumEntries(),B->getPoints()->GetNumEntries(),numElements);
	}
	else{
		numElements = A->getPoints()->GetNumEntries();
	}

	//float miOut = 0;
	float miOut = miRun((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), bins_, numElements);
	//struct cudaHistOptions *p_opt = 0;
	//float miOut = cudaMIa((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), numElements, MI_BINS, MI_BINS, p_opt, 1, true);

	*value = miOut;
}

SSD::SSD(){};

void SSD::EvalMetric(SparseScan* A, SparseScan* B, float* value){
	
	//check scans exist
	if(A == NULL || B == NULL){
		TRACE_ERROR("Two scans are required for the metric to operate");
		*value = 0;
		return;
	}

	size_t numElements;
	//check scans of same size
	if(A->getPoints()->GetNumEntries() != B->getPoints()->GetNumEntries()){
		numElements = (A->getPoints()->GetNumEntries() > B->getPoints()->GetNumEntries()) ? B->getPoints()->GetNumEntries() : A->getPoints()->GetNumEntries();
		TRACE_WARNING("Number of entries does not match, Scan A has %i, Scan B has %i, only using %i entries",A->getPoints()->GetNumEntries(),B->getPoints()->GetNumEntries(),numElements);
	}
	else{
		numElements = A->getPoints()->GetNumEntries();
	}

	float* out;
	CudaSafeCall(hipMalloc(&out, sizeof(float)*numElements));
	float* zeroEl;
	CudaSafeCall(hipMalloc(&zeroEl, sizeof(float)*numElements));

	SSDKernel<<<gridSize(numElements), BLOCK_SIZE>>>
		((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), numElements, out, zeroEl);
	CudaCheckError();

	//perform reduction
	int numThreads = 512;
	int numBlocks = ceil(((float)numElements)/((float)numThreads));
	
	float z = reduceEasy(zeroEl, numElements);
	CudaSafeCall(hipFree(zeroEl));
	float res = reduceEasy(out, numElements);
	CudaSafeCall(hipFree(out));

	res = res/(numElements-z);

	*value = res;
}

GOM::GOM(){};

void GOM::EvalMetric(SparseScan* A, SparseScan* B, float* value){
	
	*value = 0;

	//check scans exist
	if(A == NULL || B == NULL){
		TRACE_ERROR("Two scans are required for the metric to operate");
		return;
	}

	if(A->getNumCh() != GOM_DEPTH){
		TRACE_ERROR("GOM requires two channels (mag, phase) to operate and Scan A has %i", A->getNumCh());
		return;
	}
	if(B->getNumCh() != GOM_DEPTH){
		TRACE_ERROR("GOM requires two channels (mag, phase) to operate and Scan B has %i", B->getNumCh());
		return;
	}

	size_t numElements;
	//check scans of same size
	if(A->getNumPoints() != B->getNumPoints()){
		numElements = (A->getNumPoints() > B->getNumPoints()) ? B->getNumPoints() : A->getNumPoints();
		TRACE_WARNING("Number of entries does not match, Scan A has %i, Scan B has %i, only using %i entries",A->getNumPoints(),B->getNumPoints(),numElements);
	}
	else{
		numElements = A->getNumPoints();
	}

	float* phaseOut;
	float* magOut;
	CudaSafeCall(hipMalloc(&phaseOut, sizeof(float)*numElements));
	CudaSafeCall(hipMalloc(&magOut, sizeof(float)*numElements));
    
	GOMKernel<<<gridSize(numElements), BLOCK_SIZE>>>
		((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), numElements, phaseOut, magOut);
	CudaCheckError();

	//perform reduction
	int numThreads = 512;
	int numBlocks = ceil(((float)numElements)/((float)numThreads));
	
	float phaseRes = reduceEasy(phaseOut, numElements);
	CudaSafeCall(hipFree(phaseOut));
	
	float magRes = reduceEasy(magOut, numElements);
	CudaSafeCall(hipFree(magOut));
	
	float out = (phaseRes / magRes);
	
	*value = out;
}

LIV::LIV(float* avImg, size_t width, size_t height){
	avImg_ = new PointsList(avImg, (width*height), true);
}

LIV::~LIV(){
	delete avImg_;
}

void LIV::EvalMetric(SparseScan* A, SparseScan* B, float* value){

	//check scans exist
	if(A == NULL || B == NULL){
		TRACE_ERROR("Two scans are required for the metric to operate");
		*value = 0;
		return;
	}

	size_t numElements;
	//check scans of same size
	if(A->getNumPoints() != B->getNumPoints()){
		numElements = (A->getNumPoints() > B->getNumPoints()) ? B->getNumPoints() : A->getNumPoints();
		TRACE_WARNING("Number of entries does not match, Scan A has %i, Scan B has %i, only using %i entries",A->getNumPoints(),B->getNumPoints(),numElements);
	}
	else{
		numElements = A->getNumPoints();
	}

	float* out;
	CudaSafeCall(hipMalloc(&out, sizeof(float)*numElements));
	
	livValKernel<<<gridSize(numElements), BLOCK_SIZE>>>
		((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), (float*)avImg_->GetGpuPointer(), numElements, out);
	CudaCheckError();

	//perform reduction
	float outVal = reduceEasy(out, numElements);
	CudaSafeCall(hipFree(out));
	
	*value = outVal;
}
