#include "Scan.h"
#include "Kernel.h"

Scan::Scan(const size_t numDim, const size_t numCh,  const size_t* dimSize) : 
	numDim_(numDim),
	numCh_(numCh),
	dimSize_(dimSize)
{	
}

Scan::Scan(const size_t numDim, const size_t numCh,  const size_t* dimSize, PointsList* points) : 
	numDim_(numDim),
	numCh_(numCh),
	dimSize_(dimSize),
	points_(points){}

size_t Scan::getNumDim(void){
	return numDim_;
}

size_t Scan::getNumCh(void){
	return numCh_;
}

size_t Scan::getDimSize(size_t i){
	if(i >= numDim_){
		TRACE_ERROR("tried to get size of dimension %i, where only %i dimensions exist",(i+1),numDim_);
		return 0;
	}
	else {
		return dimSize_[i];
	}
}

size_t Scan::getNumPoints(void){
	size_t numPoints = 1;
		
	for( size_t i = 0; i < numDim_; i++ ){
		if(dimSize_[i] != 0){
			numPoints *= dimSize_[i];
		}
	}

	return numPoints;
}
	
PointsList* Scan::getPoints(void){
	return points_;
}

DenseImage::DenseImage(const size_t height, const size_t width, const size_t numCh, TextureList* points): 
	Scan(IMAGE_DIM ,numCh,setDimSize(width, height, numCh),points)
{
	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; 

}

//creates own copy of data
DenseImage::DenseImage(const size_t height, const size_t width, const size_t numCh, float* pointsIn):
	Scan(IMAGE_DIM ,numCh,setDimSize(width, height, numCh),NULL)
{
	TextureList* points = new TextureList(pointsIn, true, height, width, numCh);
	points_ = points;

	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; 
}

size_t* DenseImage::setDimSize(const size_t width, const size_t height, const size_t numCh){
	size_t* out = new size_t[3];
	out[0] = height;
	out[1] = width;
	out[3] = numCh;

	return out;
}

void DenseImage::d_interpolate(SparseScan* scan){
	if(!points_->IsOnGpu()){
		TRACE_WARNING("Dense image not on gpu, loading now");
		points_->AllocateGpu();
		points_->CpuToGpu();
		}
	
	//create texture
	hipChannelFormatDesc channelDescCoeff = hipCreateChannelDesc<float>();
	
	CudaSafeCall(hipBindTextureToArray(&tex, (hipArray_t)(points_->GetGpuPointer()), &channelDescCoeff));
	TextureList* texPoints = (TextureList*)points_;

	for(size_t i = 0; i < scan->getNumCh(); i++){

		DenseImageInterpolateKernel<<<gridSize(texPoints->GetHeight() * texPoints->GetWidth()) ,BLOCK_SIZE>>>	
			(texPoints->GetWidth(), texPoints->GetHeight(), (float*)scan->GetLocation()->GetGpuPointer(), (float)i, (float*)scan->getPoints()->GetGpuPointer(), scan->getDimSize(0));
		CudaCheckError();
	}
}

TextureList* DenseImage::getPoints(void){
	return (TextureList*)points_;
}


size_t* SparseScan::setDimSize(const size_t numCh, const size_t numDim, const size_t numPoints){
	size_t* out = new size_t(2);
	out[0] = numPoints;
	out[1] = numCh + numDim;

	return out;
}

size_t SparseScan::getNumPoints(void){
	return dimSize_[0];
}

float* SparseScan::GenLocation(size_t numDim, size_t* dimSize){

	size_t* iter = new size_t[numDim];

	size_t numEntries = 1;
		
	for( size_t i = 0; i < numDim; i++ ){
		iter[i] = 0;
		numEntries *= dimSize[i];
	}

	float* loc = new float[numEntries * numDim];

	size_t j = 0;
	bool run = true;

	//iterate over every point to fill in image locations
	while(run){
	
		for( size_t i = 0; i < numDim; i++ ){
			loc[j + numEntries*i] = (float)iter[i];
		}

		j++;
		iter[0]++;
		for( size_t i = 0; i < numDim; i++ ){
			if(iter[i] == 319){
				iter[i] = 319;
			}
			if(iter[i] >= dimSize[i]){
				if(i != (numDim-1)){
					iter[i+1]++;
				}
				iter[i] = 0;
			}
			else {
				run = true;
				break;
			}
			run = false;
		}
	}

	delete[] iter;

	return loc;
}

SparseScan::SparseScan(const size_t numDim, const size_t numCh,  const size_t numPoints): 
	Scan(numDim, numCh, setDimSize(numCh, numDim, numPoints),NULL)
{
	points_ = new PointsList(numPoints * numCh);
	location_ = new PointsList(numPoints * numDim);
}

SparseScan::SparseScan(const size_t numDim, const size_t numCh,  const size_t numPoints, PointsList* points, PointsList* location): 
	Scan(numDim,numCh,setDimSize(numCh, numDim, numPoints),NULL)
{	
	points_ = points;
	location_ = location;
}

//creates own copies of data
SparseScan::SparseScan(const size_t numDim, const size_t numCh,  const size_t numPoints, float* pointsIn, float* locationIn): 
	Scan(numDim,numCh,setDimSize(numCh, numDim, numPoints),NULL)
{	
	PointsList* points = new PointsList(pointsIn, numCh*numPoints, true);
	points_ = points;

	PointsList* location = new PointsList(locationIn, numDim*numPoints, true);
	location_ = location;
}

/*SparseScan::SparseScan(Scan in):
Scan(in.getNumDim(), in.getNumCh(),setDimSize(in.getNumCh(), in.getNumPoints())
{
	points_ = in.getPointsPointer();

	int i,j;

	size_t* iter = new size_t[numDim_];
		
	for( i = 0; i < numDim_; i++ ){
		iter[i] = 0;
	}

	j = 0;
	bool run = true;

	//iterate over every point to fill in image locations
	while(run){
	
		for( i = 0; i < numDim_; i++ ){
			location_[i + j*numDim_] = iter[i];
		}

		iter[0]++;
		for( i = 0; i < numDim_; i++ ){
			if(iter[i] >= dimSize_[i]){
				iter[i+1]++;
				iter[i] = 0;
			}
			else {
				break;
			}
			run = false;
		}
	}

	delete[] iter;
}

SparseScan::SparseScan(Scan in, PointsList* location):
	Scan(in.getNumDim(), in.getNumCh(),setDimSize(in.getNumDim(), in.getNumCh(), in.getNumPoints()))
{
	points_ = in.getPointsPointer();
	location_ = location;
}*/

PointsList* SparseScan::GetLocation(void){
	return location_;
}