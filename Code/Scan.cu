#include "Scan.h"

Scan::Scan(const size_t numDim, const size_t numCh,  const size_t* dimSize) : 
	numDim_(numDim),
	numCh_(numCh),
	dimSize_(dimSize)
{
	points_ = new PointsList(getNumPoints());		
}

Scan::Scan(const size_t numDim, const size_t numCh,  const size_t* dimSize, PointsList* points) : 
	numDim_(numDim),
	numCh_(numCh),
	dimSize_(dimSize),
	points_(points){}

size_t Scan::getNumDim(void){
	return numDim_;
}

size_t Scan::getNumCh(void){
	return numCh_;
}

size_t Scan::getDimSize(size_t i){
	if(i >= numDim_){
		TRACE_ERROR("tried to get size of dimension %i, where only %i dimensions exist\n",(i+1),numDim_);
		return 0;
	}
	else {
		return dimSize_[i];
	}
}

size_t Scan::getNumPoints(void){
	size_t numPoints = numCh_;
		
	for( size_t i = 0; i < numDim_; i++ ){
		numPoints *= dimSize_[i];
	}

	return numPoints;
}
	
PointsList* Scan::getPoints(void){
	return points_;
}

//dense scan points stored in a little endien (changing first dimension first) grid
DenseImage::DenseImage(const size_t height, const size_t width, const size_t numCh): 
	Scan(IMAGE_DIM ,numCh,setDimSize(width, height, numCh))
{
	points_ = new TextureList(height, width, numCh);

	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; 
}

DenseImage::DenseImage(const size_t height, const size_t width, const size_t numCh, TextureList* points): 
	Scan(IMAGE_DIM ,numCh,setDimSize(width, height, numCh))
{
	points_ = points;

	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; 
}

DenseImage::DenseImage(const size_t height, const size_t width, const size_t numCh, float* pointsIn):
	Scan(IMAGE_DIM ,numCh,setDimSize(width, height, numCh))
{
	TextureList* points = new TextureList(pointsIn, height, width, numCh);
	points_ = points;

	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; 
}

DenseImage::~DenseImage(void){
	delete points_;
}

size_t* DenseImage::setDimSize(const size_t width, const size_t height, const size_t numCh){
	size_t* out = new size_t[3];
	out[0] = height;
	out[1] = width;
	out[3] = numCh;

	return out;
}

void DenseImage::d_interpolate(SparseScan* scan){
	//create texture
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(sizeof(float),0,0,0,hipChannelFormatKindFloat);
	
	for(size_t i = 0; i < scan->getNumCh(); i++){

		hipBindTextureToArray(&tex, ((hipArray**)(points_->GetGpuPointer()))[i], &channelDesc);

		TextureList* texPoints = (TextureList*)points_;

		DenseImageInterpolateKernel<<<gridSize(texPoints->GetHeight() * texPoints->GetWidth()) ,BLOCK_SIZE>>>
			(texPoints->GetWidth(), texPoints->GetHeight(), (float*)scan->GetLocation()->GetGpuPointer(), (float*)scan->getPoints()->GetGpuPointer(), scan->getDimSize(0));
	}
}



size_t* SparseScan::setDimSize(const size_t numCh, const size_t numPoints){
	size_t* out = new size_t(2);
	out[0] = numPoints;
	out[1] = numCh;

	return out;
}

void SparseScan::GenLocation(void){

	size_t* iter = new size_t[numDim_];

	size_t numEntries = 1;
		
	for( size_t i = 0; i < numDim_; i++ ){
		iter[i] = 0;
		numEntries *= dimSize_[i];
	}

	float* loc = new float[numEntries * numDim_];

	size_t j = 0;
	bool run = true;

	//iterate over every point to fill in image locations
	while(run){
	
		for( size_t i = 0; i < numDim_; i++ ){
			loc[j + numEntries] = (float)iter[i];
		}

		j++;
		iter[0]++;
		for( size_t i = 0; i < numDim_; i++ ){
			if(iter[i] >= dimSize_[i]){
				iter[i+1]++;
				iter[i] = 0;
			}
			else {
				break;
			}
			run = false;
		}
	}

	delete[] iter;

	location_ = new PointsList(loc, numEntries * numDim_);
}

SparseScan::SparseScan(const size_t numDim, const size_t numCh,  const size_t numPoints): 
	Scan(numDim, numCh, setDimSize(numCh, numPoints))
{
	points_ = new PointsList(numPoints * numCh);
	location_ = new PointsList(numPoints * numDim);
}

SparseScan::SparseScan(const size_t numDim, const size_t numCh,  const size_t numPoints, PointsList* points, PointsList* location): 
	Scan(numDim,numCh,setDimSize(numCh,numPoints))
{	
	points_ = points;
	location_ = location;
}

SparseScan::SparseScan(const size_t numDim, const size_t numCh,  const size_t numPoints, PointsList* points): 
	Scan(numDim,numCh,setDimSize(numCh,numPoints))
{	
	points_ = points;
	GenLocation();
}

/*SparseScan::SparseScan(Scan in):
Scan(in.getNumDim(), in.getNumCh(),setDimSize(in.getNumCh(), in.getNumPoints())
{
	points_ = in.getPointsPointer();

	int i,j;

	size_t* iter = new size_t[numDim_];
		
	for( i = 0; i < numDim_; i++ ){
		iter[i] = 0;
	}

	j = 0;
	bool run = true;

	//iterate over every point to fill in image locations
	while(run){
	
		for( i = 0; i < numDim_; i++ ){
			location_[i + j*numDim_] = iter[i];
		}

		iter[0]++;
		for( i = 0; i < numDim_; i++ ){
			if(iter[i] >= dimSize_[i]){
				iter[i+1]++;
				iter[i] = 0;
			}
			else {
				break;
			}
			run = false;
		}
	}

	delete[] iter;
}

SparseScan::SparseScan(Scan in, PointsList* location):
	Scan(in.getNumDim(), in.getNumCh(),setDimSize(in.getNumDim(), in.getNumCh(), in.getNumPoints()))
{
	points_ = in.getPointsPointer();
	location_ = location;
}*/

PointsList* SparseScan::GetLocation(void){
	return location_;
}