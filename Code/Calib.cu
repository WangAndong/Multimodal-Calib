#include "Calib.h"
#include <algorithm>
#include <string> 

Calib::Calib(std::string metricType){
	checkForCUDA();
}

bool Calib::getIfPanoramic(size_t idx){
	return NULL;
}

size_t Calib::getNumPoints(size_t idx){
	return moveStore.getNumPoints(idx);
}

size_t Calib::getNumDim(size_t idx){
	return moveStore.getNumDim(idx);
}

size_t Calib::getImageDepth(size_t idx){
	return baseStore.getDepth(idx);
}

size_t Calib::getNumCh(size_t idx){
	return moveStore.getNumCh(idx);
}

size_t Calib::getNumImages(void){
	return baseStore.getNumImages();
}

size_t Calib::getImageWidth(size_t idx){
	return baseStore.getWidth(idx);
}

size_t Calib::getImageHeight(size_t idx){
	return baseStore.getHeight(idx);
}

void Calib::clearScans(void){
	moveStore.removeAllScans();
}

void Calib::clearImages(void){
	baseStore.removeAllImages();
}

void Calib::clearTforms(void){
	return;
}

void Calib::clearExtras(void){
	return;
}

void Calib::clearIndices(void){
	tformIdx.clear();
	scanIdx.clear();
}

void Calib::addScan(std::vector<thrust::host_vector<float>>& scanLIn, std::vector<thrust::host_vector<float>>& scanIIn){
	moveStore.addScan(scanLIn, scanIIn);
}

void Calib::addImage(thrust::host_vector<float>& imageIn, size_t height, size_t width, size_t depth){
	baseStore.addImage(imageIn, height, width, depth);
}

/*void Calib::addTform(thrust::host_vector<float>& tformIn, size_t tformSizeX, size_t tformSizeY){
	tformStore.addTforms(tformIn, tformSizeX, tformSizeY);
}*/

void Calib::addTform(thrust::host_vector<float>& tformIn){
	return;
}

float Calib::evalMetric(void){
	return 0;
}

void Calib::addTformIndices(std::vector<size_t>& tformsIdxIn){
	tformIdx.insert(tformIdx.end(), tformsIdxIn.begin(), tformsIdxIn.end());
}

void Calib::addScanIndices(std::vector<size_t>& scansIdxIn){
	scanIdx.insert(scanIdx.end(),scansIdxIn.begin(), scansIdxIn.end());
}

void Calib::setSSDMetric(void){
	metric = new SSD();
}

void Calib::setGOMMetric(void){
	metric = new GOM();
}

void Calib::setGOMSMetric(void){
	metric = new GOMS();
}

void Calib::setMIMetric(void){
	metric = new MI(50);
}

void Calib::setNMIMetric(void){
	metric = new NMI(50);
}


void Calib::addCameraIndices(std::vector<size_t>& cameraIdxIn){
	mexErrMsgTxt("Attempted to setup camera for use with non-camera calibration");
	return;
}

void Calib::addCamera(thrust::host_vector<float>& cameraIn, boolean panoramic){
	mexErrMsgTxt("Attempted to setup camera for use with non-camera calibration");
	return;
}

void Calib::generateImage(thrust::device_vector<float>& image, size_t width, size_t height, size_t dilate, size_t idx, bool imageColour){
	return;
}

void Calib::colourScan(float* scan, size_t idx){
	return;
}

CameraCalib::CameraCalib(std::string metricType) : Calib(metricType){}

bool CameraCalib::getIfPanoramic(size_t idx){
	return cameraStore.getPanoramic(idx);
}

void CameraCalib::clearTforms(void){
	tformStore.removeAllTforms();
}

void CameraCalib::clearExtras(void){
	cameraStore.removeAllCameras();
	return;
}

void CameraCalib::clearIndices(void){
	tformIdx.clear();
	scanIdx.clear();
	cameraIdx.clear();
}

void CameraCalib::addTform(thrust::host_vector<float>& tformIn){
	tformStore.addTforms(tformIn);
}

void CameraCalib::addCameraIndices(std::vector<size_t>& cameraIdxIn){
	cameraIdx.insert(cameraIdx.end(),cameraIdxIn.begin(), cameraIdxIn.end());
}

void CameraCalib::addCamera(thrust::host_vector<float>& cameraIn, boolean panoramic){
	cameraStore.addCams(cameraIn, panoramic);
}

float CameraCalib::evalMetric(void){

	std::vector<float> metricVal;

	if(tformIdx.size() != baseStore.getNumImages()){
		std::ostringstream err; err << "Transform index has not been correctly set up";
		mexErrMsgTxt(err.str().c_str());
		return 0;
	}
	if(cameraIdx.size() != baseStore.getNumImages()){
		std::ostringstream err; err << "Camera index has not been correctly set up";
		mexErrMsgTxt(err.str().c_str());
		return 0;
	}
	if(scanIdx.size() != baseStore.getNumImages()){
		std::ostringstream err; err << "Scan index has not been correctly set up";
		mexErrMsgTxt(err.str().c_str());
		return 0;
	}

	float out = 0;

	//hipEvent_t start, stop;
	//float time;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	

	for(size_t j = 0; j < baseStore.getNumImages(); j++){
		for(size_t i = 0; i < IMAGE_DIM; i++){
			CudaSafeCall(hipMemsetAsync(moveStore.getGLP(scanIdx[j],i),0,moveStore.getNumPoints(scanIdx[j]),moveStore.getStream(scanIdx[j])));
		}
		for(size_t i = 0; i < moveStore.getNumCh(scanIdx[j]); i++){
			CudaSafeCall(hipMemsetAsync(moveStore.getGIP(scanIdx[j],i),0,moveStore.getNumPoints(scanIdx[j]),moveStore.getStream(scanIdx[j])));
		}
 	
			//hipEventRecord(start, 0);
		tformStore.transform(&moveStore, &cameraStore, tformIdx[j], cameraIdx[j], scanIdx[j]);
			//hipEventRecord(stop, 0);hipEventSynchronize(stop);hipEventElapsedTime(&time, start, stop);mexPrintf ("Time for transform: %f ms\n", time);

			//hipEventRecord(start, 0);
		baseStore.interpolateImage(&moveStore, j, scanIdx[j], true);
			//hipEventRecord(stop, 0);hipEventSynchronize(stop);hipEventElapsedTime(&time, start, stop);mexPrintf ("Time for interpolation: %f ms\n", time);

			//hipEventRecord(start, 0);
		out += metric->evalMetric(&moveStore, scanIdx[j]);
			//hipEventRecord(stop, 0);hipEventSynchronize(stop);hipEventElapsedTime(&time, start, stop);mexPrintf ("Time for evaluation: %f ms\n", time);

	}

	return out;
}

void CameraCalib::generateImage(thrust::device_vector<float>& image, size_t width, size_t height, size_t dilate, size_t idx, bool imageColour){

	if(imageColour){
		image.resize(baseStore.getDepth(idx)*width*height);
	}
	else{
		image.resize(moveStore.getNumCh(scanIdx[idx])*width*height);
	}

	tformStore.transform(&moveStore, &cameraStore, tformIdx[idx], cameraIdx[idx], scanIdx[idx]);
	hipDeviceSynchronize();

	if(imageColour){
		baseStore.interpolateImage(&moveStore, idx, scanIdx[idx], true);
		hipDeviceSynchronize();

		for(size_t i = 0; i < baseStore.getDepth(idx); i++){
			generateOutputKernel<<<gridSize(moveStore.getNumPoints(scanIdx[idx])) ,BLOCK_SIZE>>>(
				moveStore.getGLP(scanIdx[idx],0),
				moveStore.getGLP(scanIdx[idx],1),
				moveStore.getGIP(scanIdx[idx],i),
				thrust::raw_pointer_cast(&image[width*height*i]),
				width,
				height,
				moveStore.getNumPoints(scanIdx[idx]),
				dilate);
		}
	}
	else{
		for(size_t i = 0; i < moveStore.getNumCh(scanIdx[idx]); i++){
			generateOutputKernel<<<gridSize(moveStore.getNumPoints(scanIdx[idx])) ,BLOCK_SIZE>>>(
				moveStore.getGLP(scanIdx[idx],0),
				moveStore.getGLP(scanIdx[idx],1),
				moveStore.getIP(scanIdx[idx],i),
				thrust::raw_pointer_cast(&image[width*height*i]),
				width,
				height,
				moveStore.getNumPoints(scanIdx[idx]),
				dilate);
		}
	}

	CudaCheckError();
}

void CameraCalib::colourScan(float* scan, size_t idx){
	
	moveStore.setGenIDepth(scanIdx[idx], baseStore.getDepth(idx));
	tformStore.transform(&moveStore, &cameraStore, tformIdx[idx], cameraIdx[idx], scanIdx[idx]);

	baseStore.interpolateImage(&moveStore, idx, scanIdx[idx], true);

	hipDeviceSynchronize();

	for(size_t j = 0; j < moveStore.getNumCh(idx); j++){
		hipMemcpy(&scan[j*moveStore.getNumPoints(idx)],moveStore.getIP(idx,j),moveStore.getNumPoints(idx)*sizeof(float),hipMemcpyDeviceToHost);
	}
	for(size_t j = 0; j < baseStore.getDepth(idx); j++){
		hipMemcpy(&scan[(j+moveStore.getNumCh(idx))*moveStore.getNumPoints(idx)],moveStore.getGIP(scanIdx[idx],j),moveStore.getNumPoints(idx)*sizeof(float),hipMemcpyDeviceToHost);
	}
	CudaCheckError();
}

ImageCalib::ImageCalib(std::string metricType) : Calib(metricType){}

bool ImageCalib::getIfPanoramic(size_t idx){
	return NULL;
}

void ImageCalib::clearTforms(void){
	tformStore.removeAllTforms();
}

void ImageCalib::clearExtras(void){
	return;
}

void ImageCalib::clearIndices(void){
	tformIdx.clear();
	scanIdx.clear();
}

void ImageCalib::addTform(thrust::host_vector<float>& tformIn){
	tformStore.addTforms(tformIn);
}

float ImageCalib::evalMetric(void){

	std::vector<float> metricVal;

	if(tformIdx.size() != baseStore.getNumImages()){
		std::ostringstream err; err << "Transform index has not been correctly set up";
		mexErrMsgTxt(err.str().c_str());
		return 0;
	}
	if(scanIdx.size() != baseStore.getNumImages()){
		std::ostringstream err; err << "Scan index has not been correctly set up";
		mexErrMsgTxt(err.str().c_str());
		return 0;
	}

	float out = 0;

	//hipEvent_t start, stop;
	//float time;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	
	
	for(size_t j = 0; j < baseStore.getNumImages(); j++){
				
		for(size_t i = 0; i < IMAGE_DIM; i++){
			CudaSafeCall(hipMemsetAsync(moveStore.getGLP(scanIdx[j],i),0,sizeof(float)*moveStore.getNumPoints(scanIdx[j]),moveStore.getStream(scanIdx[j])));
		}
		for(size_t i = 0; i < moveStore.getNumCh(scanIdx[j]); i++){
			CudaSafeCall(hipMemsetAsync(moveStore.getGIP(scanIdx[j],i),0,sizeof(float)*moveStore.getNumPoints(scanIdx[j]),moveStore.getStream(scanIdx[j])));
		}

			//hipEventRecord(start, 0);
		tformStore.transform(&moveStore, &noCamera, tformIdx[j], NULL, scanIdx[j]);
			//hipEventRecord(stop, 0);hipEventSynchronize(stop);hipEventElapsedTime(&time, start, stop);mexPrintf ("Time for transform: %f ms\n", time);
		//hipStreamSynchronize(moveStore.getStream(scanIdx[j]));
			//hipEventRecord(start, 0);
		baseStore.interpolateImage(&moveStore, j, scanIdx[j], true);
			//hipEventRecord(stop, 0);hipEventSynchronize(stop);hipEventElapsedTime(&time, start, stop);mexPrintf ("Time for interpolation: %f ms\n", time);

			//hipEventRecord(start, 0);
		out += metric->evalMetric(&moveStore, scanIdx[j]);
			//hipEventRecord(stop, 0);hipEventSynchronize(stop);hipEventElapsedTime(&time, start, stop);mexPrintf ("Time for evaluation: %f ms\n", time);
	}

	return out;
}

void ImageCalib::generateImage(thrust::device_vector<float>& image, size_t width, size_t height, size_t dilate, size_t idx, bool imageColour){

	if(imageColour){
		image.resize(baseStore.getDepth(idx)*width*height);
		moveStore.setGenIDepth(scanIdx[idx], baseStore.getDepth(idx));
	}
	else{
		image.resize(moveStore.getNumCh(scanIdx[idx])*width*height);
	}

	tformStore.transform(&moveStore, &noCamera, tformIdx[idx], NULL, scanIdx[idx]);
	hipDeviceSynchronize();

	if(imageColour){
		baseStore.interpolateImage(&moveStore, idx, scanIdx[idx], true);
		hipDeviceSynchronize();

		for(size_t i = 0; i < baseStore.getDepth(idx); i++){
			
			generateOutputKernel<<<gridSize(moveStore.getNumPoints(scanIdx[idx])) ,BLOCK_SIZE>>>(
				moveStore.getGLP(scanIdx[idx],0),
				moveStore.getGLP(scanIdx[idx],1),
				moveStore.getGIP(scanIdx[idx],i),
				thrust::raw_pointer_cast(&image[width*height*i]),
				width,
				height,
				moveStore.getNumPoints(scanIdx[idx]),
				dilate);
		}
	}
	else{
		for(size_t i = 0; i < moveStore.getNumCh(scanIdx[idx]); i++){
			generateOutputKernel<<<gridSize(moveStore.getNumPoints(scanIdx[idx])) ,BLOCK_SIZE>>>(
				moveStore.getGLP(scanIdx[idx],0),
				moveStore.getGLP(scanIdx[idx],1),
				moveStore.getIP(scanIdx[idx],i),
				thrust::raw_pointer_cast(&image[width*height*i]),
				width,
				height,
				moveStore.getNumPoints(scanIdx[idx]),
				dilate);
		}
	}

	CudaCheckError();
}

void ImageCalib::colourScan(float* scan, size_t idx){
	
	moveStore.setGenIDepth(scanIdx[idx], baseStore.getDepth(idx));

	tformStore.transform(&moveStore, &noCamera, tformIdx[idx], NULL, scanIdx[idx]);

	baseStore.interpolateImage(&moveStore, idx, scanIdx[idx], true);

	hipDeviceSynchronize();

	for(size_t j = 0; j < moveStore.getNumCh(idx); j++){
		hipMemcpy(&scan[j*moveStore.getNumPoints(idx)],moveStore.getIP(idx,j),moveStore.getNumPoints(idx),hipMemcpyDeviceToHost);
	}
	for(size_t j = 0; j < baseStore.getDepth(idx); j++){
		hipMemcpy(&scan[(j+moveStore.getNumCh(idx))*moveStore.getNumPoints(idx)],moveStore.getGIP(scanIdx[idx],j),moveStore.getNumPoints(idx),hipMemcpyDeviceToHost);
	}
	CudaCheckError();
}


