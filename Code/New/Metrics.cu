#include "hip/hip_runtime.h"
#include "Metrics.h"
#include "Kernels.h"

float Metric::evalMetric(std::vector<float*>& gen, ScanList scan, size_t index, hipStream_t stream){
	mexErrMsgTxt("No metric has been specified");
	return 0;
}

/*MI::MI(size_t bins):
	bins_(bins){
}

void MI::evalMetric(std::vector<float*> A, std::vector< thrust::device_vector<float>> B, hipStream_t stream){
	
	//check scans exist
	if(A == NULL || B == NULL){
		TRACE_ERROR("Two scans are required for the metric to operate");
		*value = 0;
		return;
	}

	size_t numElements;
	//check scans of same size
	if(A->getPoints()->GetNumEntries() != B->getPoints()->GetNumEntries()){
		numElements = (A->getPoints()->GetNumEntries() > B->getPoints()->GetNumEntries()) ? B->getPoints()->GetNumEntries() : A->getPoints()->GetNumEntries();
		TRACE_WARNING("Number of entries does not match, Scan A has %i, Scan B has %i, only using %i entries",A->getPoints()->GetNumEntries(),B->getPoints()->GetNumEntries(),numElements);
	}
	else{
		numElements = A->getPoints()->GetNumEntries();
	}

	//float miOut = 0;
	float miOut = miRun((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), bins_, numElements, stream);
	//struct cudaHistOptions *p_opt = 0;
	//float miOut = cudaMIa((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), numElements, MI_BINS, MI_BINS, p_opt, 1, true);

	*value = miOut;
}*/

SSD::SSD(){};

float SSD::evalMetric(std::vector<float*>& gen, ScanList scan, size_t index, hipStream_t stream){
	
	if((gen.size() != 1) || (scan.getNumCh(index) != 1)){
		mexErrMsgTxt("SSD metric can only accept a single intensity channel");
	}
	SSDKernel<<<gridSize(scan.getNumPoints(index)), BLOCK_SIZE, 0, stream>>>
		(gen[0], scan.getIP(index,0), scan.getNumPoints(index));
	CudaCheckError();

	//perform reduction
	return thrust::reduce(&gen[0][0], &gen[0][scan.getNumPoints(index)-1], 0.0f);
}
/*
GOM::GOM(){};

void GOM::evalMetric(std::vector<float*> A, std::vector< thrust::device_vector<float>> B, hipStream_t stream){
	
	*value = 0;

	//check scans exist
	if(A == NULL || B == NULL){
		TRACE_ERROR("Two scans are required for the metric to operate");
		return;
	}

	if(A->getNumCh() != GOM_DEPTH){
		TRACE_ERROR("GOM requires two channels (mag, phase) to operate and Scan A has %i", A->getNumCh());
		return;
	}
	if(B->getNumCh() != GOM_DEPTH){
		TRACE_ERROR("GOM requires two channels (mag, phase) to operate and Scan B has %i", B->getNumCh());
		return;
	}

	size_t numElements;
	//check scans of same size
	if(A->getNumPoints() != B->getNumPoints()){
		numElements = (A->getNumPoints() > B->getNumPoints()) ? B->getNumPoints() : A->getNumPoints();
		TRACE_WARNING("Number of entries does not match, Scan A has %i, Scan B has %i, only using %i entries",A->getNumPoints(),B->getNumPoints(),numElements);
	}
	else{
		numElements = A->getNumPoints();
	}

	float* phaseOut;
	float* magOut;
	CudaSafeCall(hipMalloc(&phaseOut, sizeof(float)*numElements));
	CudaSafeCall(hipMalloc(&magOut, sizeof(float)*numElements));
    
	GOMKernel<<<gridSize(numElements), BLOCK_SIZE, 0, *stream>>>
		((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), numElements, phaseOut, magOut);
	CudaCheckError();

	//perform reduction
	int numThreads = 512;
	int numBlocks = ceil(((float)numElements)/((float)numThreads));
	
	float phaseRes = reduceEasy(phaseOut, numElements);
	CudaSafeCall(hipFree(phaseOut));
	
	float magRes = reduceEasy(magOut, numElements);
	CudaSafeCall(hipFree(magOut));
	
	float out = (phaseRes / magRes);
	
	*value = out;
}

LIV::LIV(float* avImg, size_t width, size_t height){
	avImg_ = new PointsList(avImg, (width*height), true);
}

LIV::~LIV(){
	delete avImg_;
}

void LIV::evalMetric(std::vector<float*> A, std::vector< thrust::device_vector<float>> B, hipStream_t stream){

	//check scans exist
	if(A == NULL || B == NULL){
		TRACE_ERROR("Two scans are required for the metric to operate");
		*value = 0;
		return;
	}

	size_t numElements;
	//check scans of same size
	if(A->getNumPoints() != B->getNumPoints()){
		numElements = (A->getNumPoints() > B->getNumPoints()) ? B->getNumPoints() : A->getNumPoints();
		TRACE_WARNING("Number of entries does not match, Scan A has %i, Scan B has %i, only using %i entries",A->getNumPoints(),B->getNumPoints(),numElements);
	}
	else{
		numElements = A->getNumPoints();
	}

	float* out;
	CudaSafeCall(hipMalloc(&out, sizeof(float)*numElements));
	
	livValKernel<<<gridSize(numElements), BLOCK_SIZE, 0, *stream>>>
		((float*)A->getPoints()->GetGpuPointer(), (float*)B->getPoints()->GetGpuPointer(), (float*)avImg_->GetGpuPointer(), numElements, out);
	CudaCheckError();

	//perform reduction
	float outVal = reduceEasy(out, numElements);
	CudaSafeCall(hipFree(out));
	
	*value = outVal;
}*/
