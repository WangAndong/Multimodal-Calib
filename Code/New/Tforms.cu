#include "Tforms.h"
#include "ScanList.h"
#include "ImageList.h"
#include "Kernels.h"

void Tforms::addTforms(thrust::device_vector<float> tformDIn, size_t tformSizeX, size_t tformSizeY){
	if(tformDIn.size() != (tformSizeX*tformSizeY)){
		std::cerr << "Error input tform matricies must be same size as given dimensions in size. Returning without setting\n";
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = tformSizeX;
	tformD.back().tformSizeY = tformSizeY;
}

void Tforms::addTforms(thrust::host_vector<float> tformDIn, size_t tformSizeX, size_t tformSizeY){
	if(tformDIn.size() != (tformSizeX*tformSizeY)){
		std::cerr << "Error input tform matricies must be same size as given dimensions in size. Returning without setting\n";
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = tformSizeX;
	tformD.back().tformSizeY = tformSizeY;
}

void Tforms::removeAllTforms(void){
	tformD.clear();
}

float* Tforms::getTformP(size_t idx){
	if(tformD.size() > idx){
		std::cerr << "Cannot get pointer to element " << idx << " as only " << tformD.size() << " elements exist. Returning NULL\n";
		return NULL;
	}
	return thrust::raw_pointer_cast(&(tformD[idx].tform[0]));
}

size_t Tforms::getTformSize(size_t idx){
	if(tformD.size() > idx){
		std::cerr << "Cannot get element " << idx << " as only " << tformD.size() << " elements exist. Returning 0\n";
		return 0;
	}
	return (tformD[idx].tformSizeX * tformD[idx].tformSizeY);
}

void CameraTforms::addTforms(thrust::device_vector<float> tformDIn){
	if(tformDIn.size() != 16){
		std::cerr << "Error input tform matricies must be same size as given dimensions in size. Returning without setting\n";
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 4;
	tformD.back().tformSizeY = 4;
}

void CameraTforms::addTforms(thrust::host_vector<float> tformDIn){
	if(tformDIn.size() != 16){
		std::cerr << "Error input tform matricies must be same size as given dimensions in size. Returning without setting\n";
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 4;
	tformD.back().tformSizeY = 4;
}

void CameraTforms::transform(ScanList* scansIn, std::vector<float*> locOut, Cameras* cam, size_t tformIdx, size_t camIdx, size_t scanIdx, hipStream_t stream){

	CameraTransformKernel<<<gridSize(scansIn->getNumPoints(scanIdx)), BLOCK_SIZE, 0, stream>>>(
		this->getTformP(tformIdx),
		cam->getCamP(camIdx),
		cam->getPanoramic(camIdx),
		scansIn->getLP(scanIdx,0),
		scansIn->getLP(scanIdx,1),
		scansIn->getLP(scanIdx,2),
		scansIn->getNumPoints(scanIdx),
		locOut[0],
		locOut[1]);

	CudaCheckError();
}

void AffineTforms::addTforms(thrust::host_vector<float> tformDIn){
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 3;
	tformD.back().tformSizeY = 3;
}

void AffineTforms::addTforms(thrust::device_vector<float> tformDIn){
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 3;
	tformD.back().tformSizeY = 3;
}

void AffineTform::transform(SparseScan* in, SparseScan** out, hipStream_t* stream){

	delete *out;
	*out = new SparseScan(in->getNumDim(), 0, in->getNumPoints());
	(*out)->GetLocation()->AllocateGpu();

	if(in->getNumDim() != AFFINE_DIM){
		TRACE_ERROR("affine transform can only operate on a 2d input, returning untransformed points");
		CudaSafeCall(hipMemcpy((*out)->GetLocation()->GetGpuPointer(), in->GetLocation()->GetGpuPointer(), in->getNumDim()*in->getNumPoints()*sizeof(float), hipMemcpyDeviceToDevice));
		return;
	}

	AffineTransformKernel<<<gridSize(in->getDimSize(0)), BLOCK_SIZE, 0, *stream>>>(d_tform_, (float*)in->GetLocation()->GetGpuPointer(), (float*)(*out)->GetLocation()->GetGpuPointer(), in->getDimSize(0));
	 //CudaSafeCall(hipMemcpy(out->GetLocation()->GetGpuPointer(), in->GetLocation()->GetGpuPointer(), in->getNumPoints()*sizeof(float), hipMemcpyDeviceToDevice));
}
