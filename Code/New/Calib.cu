#include "Calib.h"
#include <algorithm>
#include <string> 

Calib::Calib(std::string tformType, std::string metricType){
	checkForCUDA();

	std::transform(tformType.begin(), tformType.end(), tformType.begin(), ::tolower);
	std::transform(tformType.begin(), tformType.end(), tformType.begin(), ::tolower);

	if(tformType == "affine"){
		tformStore = new AffineTforms;
	}
	else if(tformType == "camera"){
		tformStore = new CameraTforms;
	}
	else{
		std::cerr << "Error unrecognized tform " << tformType << ". Options are affine or camera, defaulting to camera";
		tformStore = new CameraTforms;
	}

	moveStore = new ScanList();
	baseStore = new ImageList();

}

void Calib::clearScans(void){
	moveStore->removeAllScans();
}

void Calib::clearImages(void){
	baseStore->removeAllImages();
}

void Calib::clearTforms(void){
	tformStore->removeAllTforms();
}

void Calib::clearEverything(void){
	clearScans();
	clearImages();
	clearTforms();
}

void Calib::addScan(std::vector<thrust::host_vector<float>> scanLIn, std::vector<thrust::host_vector<float>> scanIIn){
	if((moveStore->getNumPoints() != 0) && ((moveStore->getNumDim() != scanLIn.size()) || moveStore->getNumCh() != scanIIn.size())){
		std::cerr << "Number of dimensions and channels must match scans already set, returning without setting\n";
		return;
	}
	moveStore->addScan(scanLIn, scanIIn);
}

void Calib::addImage(thrust::host_vector<float> imageIn, size_t height, size_t width, size_t depth, size_t tformIdx, size_t scanIdx){
	baseStore->addImage(imageIn, height, width, depth, tformIdx, scanIdx);
}

void Calib::addImage(thrust::host_vector<float> imageIn, size_t height, size_t width, size_t depth, size_t tformIdx, size_t scanIdx){
	baseStore->addImage(imageIn, height, width, depth, tformIdx, scanIdx);
}

void Calib::addTform(thrust::host_vector<float> tformIn){
	if(tformStore->getTformSize() != tformIn.size()){
		std::cerr << "Tform must match size of initilized tforms, returning without setting\n";
		return;
	}
	tformStore->addTforms(tformIn);
}

float Calib::evalMetric(void){

	std::vector<std::vector<float*>> genL;
	std::vector<std::vector<float*>> genI;

	std::vector<float> metricVal;

	std::vector<hipStream_t> streams;

	size_t genLength = 0;
	for(size_t i = 0; i < moveStore->getNumScans(); i+= (genLength+1)){
		genLength = allocateGenMem(moveStore, baseStore, genL, genI, i);
		
		streams.resize(genLength-i);
		for(size_t j = 0; j < streams.size(); j++){
			hipStreamCreate ( &streams[j]);
			tformStore->transform(moveStore, genL[j], baseStore->getTformIdx(i+j), ((CameraTforms*)tformStore)->getCameraIdx(i+j), baseStore->getScanIdx(i+j), streams[j]);
		}
	}
}

size_t Calib::allocateGenMem(ScanList* points, ImageList* images, std::vector<std::vector<float*>> genL, std::vector<std::vector<float*>> genI, size_t startIdx){
	
	hipError_t err = hipSuccess;
	size_t i;

	genL.resize(images->getNumImages());
	genI.resize(images->getNumImages());

	for(i = startIdx; i < images->getNumImages(); i++){

		genL[i].resize(IMAGE_DIM);
		for(size_t j = 0; j < IMAGE_DIM; j++){
			hipError_t currentErr = hipMalloc(&genL[i][j], sizeof(float)*points->getNumPoints(images->getScanIdx(i)));
			if(currentErr != hipSuccess){
				err = hipErrorOutOfMemory;
				break;
			}
		}
		genI[i].resize(images->getDepth(i));
		for(size_t j = 0; j < images->getDepth(i); j++){
			hipError_t currentErr = hipMalloc(&genI[i][j], sizeof(float)*points->getNumPoints(images->getScanIdx(i)));
			if(currentErr != hipSuccess){
				err = hipErrorOutOfMemory;
				break;
			}
		}

		if(err == hipErrorOutOfMemory){
			for(size_t j = 0; j < IMAGE_DIM; j++){
				hipFree(&genL[i][j]);
			}
			for(size_t j = 0; j < images->getDepth(i); j++){
				hipFree(&genI[i][j]);
			}
			break;
		}
	}

	return i;
}

