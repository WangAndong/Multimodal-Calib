#include "Calib.h"
#include <algorithm>
#include <string> 

Calib::Calib(std::string metricType){
	checkForCUDA();
	/*
	std::transform(metricType.begin(), metricType.end(), metricType.begin(), ::tolower);

	if(tformType == "affine"){
		tformStore = new AffineTforms;
	}
	else if(tformType == "camera"){
		tformStore = new CameraTforms;
	}
	else{
		std::cerr << "Error unrecognized tform " << tformType << ". Options are affine or camera, defaulting to camera";
		tformStore = new CameraTforms;
	}

	moveStore = new ScanList();
	baseStore = new ImageList();
	*/
}

void Calib::clearScans(void){
	moveStore->removeAllScans();
}

void Calib::clearImages(void){
	baseStore->removeAllImages();
}

void Calib::clearTforms(void){
	tformStore->removeAllTforms();
}

void Calib::clearExtras(void){
	return;
}

void Calib::clearEverything(void){
	clearScans();
	clearImages();
	clearTforms();
}

void Calib::addScan(std::vector<thrust::host_vector<float>> scanLIn, std::vector<thrust::host_vector<float>> scanIIn){
	moveStore->addScan(scanLIn, scanIIn);
}

void Calib::addImage(thrust::host_vector<float> imageIn, size_t height, size_t width, size_t depth, size_t tformIdxIn, size_t scanIdxIn){
	tformIdx.push_back(tformIdxIn);
	scanIdx.push_back(scanIdxIn);
	baseStore->addImage(imageIn, height, width, depth);
}

void Calib::addTform(thrust::host_vector<float> tformIn, size_t tformSizeX, size_t tformSizeY){
	tformStore->addTforms(tformIn, tformSizeX, tformSizeY);
}

void Calib::addCamera(thrust::host_vector<float> cameraIn, boolean panoramic){};

float Calib::evalMetric(void){
	return 0;
}

size_t Calib::allocateGenMem(ScanList* points, ImageList* images, std::vector<std::vector<float*>> genL, std::vector<std::vector<float*>> genI, size_t startIdx){
	
	hipError_t err = hipSuccess;
	size_t i;

	genL.resize(images->getNumImages());
	genI.resize(images->getNumImages());

	for(i = startIdx; i < images->getNumImages(); i++){

		genL[i].resize(IMAGE_DIM);
		for(size_t j = 0; j < IMAGE_DIM; j++){
			hipError_t currentErr = hipMalloc(&genL[i][j], sizeof(float)*points->getNumPoints(scanIdx[i]));
			if(currentErr != hipSuccess){
				err = hipErrorOutOfMemory;
				break;
			}
		}
		genI[i].resize(images->getDepth(i));
		for(size_t j = 0; j < images->getDepth(i); j++){
			hipError_t currentErr = hipMalloc(&genI[i][j], sizeof(float)*points->getNumPoints(scanIdx[i]));
			if(currentErr != hipSuccess){
				err = hipErrorOutOfMemory;
				break;
			}
		}

		if(err == hipErrorOutOfMemory){
			for(size_t j = 0; j < IMAGE_DIM; j++){
				hipFree(&genL[i][j]);
			}
			for(size_t j = 0; j < images->getDepth(i); j++){
				hipFree(&genI[i][j]);
			}
			break;
		}
	}

	return i;
}

CameraCalib::CameraCalib(std::string metricType) : Calib(metricType){}

void CameraCalib::addImage(thrust::host_vector<float> imageIn, size_t height, size_t width, size_t depth, size_t tformIdxIn, size_t scanIdxIn, size_t cameraIdxIn){
	tformIdx.push_back(tformIdxIn);
	scanIdx.push_back(scanIdxIn);
	cameraIdx.push_back(cameraIdxIn);
	baseStore->addImage(imageIn, height, width, depth);
}

void CameraCalib::addCamera(thrust::host_vector<float> cameraIn, boolean panoramic){
	cameraStore->addCams(cameraIn, panoramic);
}

float CameraCalib::evalMetric(void){

	std::vector<std::vector<float*>> genL;
	std::vector<std::vector<float*>> genI;

	std::vector<float> metricVal;

	std::vector<hipStream_t> streams;

	size_t genLength = 0;
	for(size_t i = 0; i < moveStore->getNumScans(); i+= (genLength+1)){
		genLength = allocateGenMem(moveStore, baseStore, genL, genI, i);
		
		streams.resize(genLength-i);
		for(size_t j = 0; j < streams.size(); j++){
			hipStreamCreate ( &streams[j]);
			tformStore->transform(moveStore, genL[j], cameraStore, tformIdx[i+j], cameraIdx[i+j], scanIdx[i+j], streams[j]);

		}
	}

	return 0;
}

