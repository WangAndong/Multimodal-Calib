#include "hip/hip_runtime.h"
/*
 Copyright Ramtin Shams (hereafter referred to as 'the author'). All rights 
 reserved. **Citation required in derived works or publications** 
 
 NOTICE TO USER:   
 
 Users and possessors of this source code are hereby granted a nonexclusive, 
 royalty-free license to use this source code for non-commercial purposes only, 
 as long as the author is appropriately acknowledged by inclusion of this 
 notice in derived works and citation of appropriate publication(s) listed 
 at the end of this notice in any derived works or publications that use 
 or have benefited from this source code in its entirety or in part.
   
 
 THE AUTHOR MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 IMPLIED WARRANTY OF ANY KIND.  THE AUTHOR DISCLAIMS ALL WARRANTIES WITH 
 REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 OR PERFORMANCE OF THIS SOURCE CODE.  
 
 Relevant publication(s):
	@inproceedings{Shams_ICSPCS_2007,
		author        = "R. Shams and R. A. Kennedy",
		title         = "Efficient Histogram Algorithms for {NVIDIA} {CUDA} Compatible Devices",
		booktitle     = "Proc. Int. Conf. on Signal Processing and Communications Systems ({ICSPCS})",
		address       = "Gold Coast, Australia",
		month         = dec,
		year          = "2007",
		pages         = "418-422",
	}

	@inproceedings{Shams_DICTA_2007a,
		author        = "R. Shams and N. Barnes",
		title         = "Speeding up Mutual Information Computation Using {NVIDIA} {CUDA} Hardware",
		booktitle     = "Proc. Digital Image Computing: Techniques and Applications ({DICTA})",
		address       = "Adelaide, Australia",
		month         = dec,
		year          = "2007",
		pages         = "555-560",
		doi           = "10.1109/DICTA.2007.4426846",
	};
*/

// includes, system
#include <stdlib.h>
#include <tchar.h>
#include <stdarg.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "cutil.h"
#include <hip/driver_types.h>
#include "cuda_basics.h"
#include "cuda_hist.h"
#include "cuda_mi.h"

// includes, kernels
#include "gpu_mi.cu"

__global__ void cudaHistRemoveZeros(float* d_hist, int xbins, int ybins, int* length){
	
	int i;

	for(i = 0; i < xbins; i++){
		length[0] -= (int)d_hist[i];
		d_hist[i] = 0;
	}
	for(i = xbins; i < xbins*ybins; i+=xbins){
		length[0] -= (int)d_hist[i];
		d_hist[i] = 0;
	}
}

extern "C" float cudaMIa(float *src1, float *src2, int length, int xbins, int ybins, cudaHistOptions *p_options /*= NULL*/, bool device /*= false*/, bool incZeros /*= false*/)
{
	dim3 grid, block;
	int size = length * sizeof(float);
	//Device memory pointers
	float *d_src1, *d_src2, *d_hist, *d_rows, *d_cols;

	if (!device)
	{
		//Allocate data on the device
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src1, size));
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src2, size));

		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src1, src1, size, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_src2, src2, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src1 = src1; d_src2 = src2;
	}

    CUDA_SAFE_CALL(hipDeviceSynchronize());														

	CUDA_SAFE_CALL(hipMalloc((void**) &d_hist, xbins * ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_rows, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_rows, 0, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cols, xbins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_cols, 0, xbins * sizeof(float)));

	cudaHist2Da(d_src1, d_src2, d_hist, length, xbins, ybins, p_options, true);					//No need to initialize d_hist, it will be done by cudaHist

	if(!incZeros){

		int* lengthD;
		CUDA_SAFE_CALL(hipMalloc((void**) &lengthD, sizeof(int)));
		CUDA_SAFE_CALL(hipMemcpy(lengthD, &length, sizeof(int), hipMemcpyHostToDevice));

		cudaHistRemoveZeros<<<1,1>>>(d_hist, xbins, ybins, lengthD);
	
		CUDA_SAFE_CALL(hipMemcpy(&length, lengthD, sizeof(int), hipMemcpyDeviceToHost));

		CUDA_SAFE_CALL(hipFree(lengthD));
	}

	int totalSamples = length;
	//Caculate MI from the joint histogram
	// H(X,Y)
	float Hxy = cudaEntropy(d_hist, xbins * ybins, true) / totalSamples + logf((float)totalSamples);

	// H(X)
	cudaSumAlongCols(d_hist, d_cols, xbins, ybins, true);
	float Hx = cudaEntropy(d_cols, xbins, true) / totalSamples + logf((float)totalSamples);

	// H(Y)
	cudaSumAlongRows(d_hist, d_rows, xbins, ybins, true);
	float Hy = cudaEntropy(d_rows, ybins, true) / totalSamples + logf((float)totalSamples);

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	if (!device)
	{
		CUDA_SAFE_CALL(hipFree(d_src1));
		CUDA_SAFE_CALL(hipFree(d_src2));
	}

	CUDA_SAFE_CALL(hipFree(d_hist));
	CUDA_SAFE_CALL(hipFree(d_rows));
	CUDA_SAFE_CALL(hipFree(d_cols));

	return ((Hx + Hy) / Hxy);
}

extern "C" float cudaMIb(float *src1, float *src2, int length, int xbins, int ybins, cudaHistOptions *p_options /*= NULL*/, bool device /*= false*/)
{
	dim3 grid, block;
	int size = length * sizeof(float);
	//Device memory pointers
	float *d_src1, *d_src2, *d_hist, *d_rows, *d_cols;

	if (!device)
	{
		//Allocate data on the device
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src1, size));
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src2, size));

		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src1, src1, size, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_src2, src2, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src1 = src1; d_src2 = src2;
	}

    CUDA_SAFE_CALL(hipDeviceSynchronize());														

	CUDA_SAFE_CALL(hipMalloc((void**) &d_hist, xbins * ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_rows, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_rows, 0, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cols, xbins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_cols, 0, xbins * sizeof(float)));

	cudaHist2Db(d_src1, d_src2, d_hist, length, xbins, ybins, p_options, true);					//No need to initialize d_hist, it will be done by cudaHist

	int totalSamples = length;
	//Caculate MI from the joint histogram
	// H(X,Y)
	float Hxy = cudaEntropy(d_hist, xbins * ybins, true) / totalSamples + logf((float)totalSamples);

	// H(X)
	cudaSumAlongCols(d_hist, d_cols, xbins, ybins, true);
	float Hx = cudaEntropy(d_cols, xbins, true) / totalSamples + logf((float)totalSamples);

	// H(Y)
	cudaSumAlongRows(d_hist, d_rows, xbins, ybins, true);
	float Hy = cudaEntropy(d_rows, ybins, true) / totalSamples + logf((float)totalSamples);

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	if (!device)
	{
		CUDA_SAFE_CALL(hipFree(d_src1));
		CUDA_SAFE_CALL(hipFree(d_src2));
	}

	CUDA_SAFE_CALL(hipFree(d_hist));
	CUDA_SAFE_CALL(hipFree(d_rows));
	CUDA_SAFE_CALL(hipFree(d_cols));

	return Hx + Hy - Hxy;
}

extern "C" float cudaMI_Approx(float *src1, float *src2, int length, int xbins, int ybins, cudaHistOptions *p_options /*= NULL*/, bool device /*= false*/)
{
	dim3 grid, block;
	int size = length * sizeof(float);
	//Device memory pointers
	float *d_src1, *d_src2, *d_hist, *d_rows, *d_cols;

	if (!device)
	{
		//Allocate data on the device
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src1, size));
		CUDA_SAFE_CALL(hipMalloc((void**) &d_src2, size));

		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src1, src1, size, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_src2, src2, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src1 = src1; d_src2 = src2;
	}

    CUDA_SAFE_CALL(hipDeviceSynchronize());															

	CUDA_SAFE_CALL(hipMalloc((void**) &d_hist, xbins * ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_rows, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_rows, 0, ybins * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_cols, xbins * sizeof(float)));
	CUDA_SAFE_CALL(hipMemset(d_cols, 0, xbins * sizeof(float)));

	cudaHist2D_Approx(d_src1, d_src2, d_hist, length, xbins, ybins, p_options, true);					//No need to initialize d_hist, it will be done by cudaHist

	int totalSamples = (int) cudaSum(d_hist, xbins * ybins, true);
	//Caculate MI from the joint histogram
	// H(X,Y)
	float Hxy = cudaEntropy(d_hist, xbins * ybins, true) / totalSamples + logf((float)totalSamples);

	// H(X)
	cudaSumAlongCols(d_hist, d_cols, xbins, ybins, true);
	float Hx = cudaEntropy(d_cols, xbins, true) / totalSamples + logf((float)totalSamples);

	// H(Y)
	cudaSumAlongRows(d_hist, d_rows, xbins, ybins, true);
	float Hy = cudaEntropy(d_rows, ybins, true) / totalSamples + logf((float)totalSamples);

	if (!device)
	{
		CUDA_SAFE_CALL(hipFree(d_src1));
		CUDA_SAFE_CALL(hipFree(d_src2));
	}

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(hipFree(d_hist));
	CUDA_SAFE_CALL(hipFree(d_rows));
	CUDA_SAFE_CALL(hipFree(d_cols));

	return Hx + Hy - Hxy;
}

extern "C" float cudaEntropy(float *src, int length, bool device /*= false*/)
{
	float *d_src, *d_dst;
	int size = length * sizeof(float);
	if (!device)
	{
		CUDA_SAFE_CALL(hipMalloc((void**)&d_src, size));
		//Copy src data to device memory
		CUDA_SAFE_CALL(hipMemcpy(d_src, src, size, hipMemcpyHostToDevice));
	}
	else
	{
		d_src = src;
	}
	CUDA_SAFE_CALL(hipMalloc((void**)&d_dst, size));

	cudaEntropyUnary(d_src, d_dst, length, true);
	float res = cudaSum(d_dst, length, true);

	if (!device)																				
		CUDA_SAFE_CALL(hipFree(d_src));
	CUDA_SAFE_CALL(hipFree(d_dst));

	return res;
}

CUDA_UNARY(EntropyUnary);
