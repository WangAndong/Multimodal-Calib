#include "hip/hip_runtime.h"
#include "Kernel.h"

__global__ void DenseImageInterpolateKernel(const size_t width, const size_t height, const float* locIn, float* valsOut, const size_t numPoints){
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= numPoints){
		valsOut[i] = 0.0f;
		return;
	}

	bool inside =
		-0.5f < locIn[i] && locIn[i] < (width - 0.5f) &&
		-0.5f < locIn[i + numPoints] && locIn[i + numPoints] < (height - 0.5f);

	if (!inside){
		valsOut[i] = 0.0f;
	}
	else{
		//valsOut[i] = cubicTex2D(tex, locIn[i]+0.5f, locIn[i + numPoints]+0.5f);
	}
}

__global__ void AffineTransformKernel(const float* tform, const float* pointsIn, float* pointsOut, const size_t numPoints){
	
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= numPoints){
		return;
	}

	//make it a bit clearer which are x and y points
	const float xIn = pointsIn[i];
	const float yIn = pointsIn[i + numPoints];

	//transform points
	float xOut = xIn*tform[0] + yIn*tform[3] + tform[6];
	float yOut = xIn*tform[1] + yIn*tform[4] + tform[7];

	pointsOut[i] = xOut;
	pointsOut[i + numPoints] = yOut;

}

__global__ void CameraTransformKernel(const float* tform, const float* cam, const float* pointsIn, float* pointsOut, const size_t numPoints, const bool panoramic){
	
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= numPoints){
		return;
	}

	const float xIn = pointsIn[i + 0*numPoints];
	const float yIn = pointsIn[i + 1*numPoints];
	const float zIn = pointsIn[i + 2*numPoints];

	//transform points
	float x = xIn*tform[0] + yIn*tform[4] + zIn*tform[8] + tform[12];
	float y = xIn*tform[1] + yIn*tform[5] + zIn*tform[9] + tform[13];
	float z = xIn*tform[2] + yIn*tform[6] + zIn*tform[10] + tform[14];

	if((z <= 0) && !panoramic){
		x = -1;
		y = -1;
	}
	else{

		//apply projective camera matrix
		x = cam[0]*x + cam[3]*y + cam[6]*z + cam[9];
		y = cam[1]*x + cam[4]*y + cam[7]*z + cam[10];
		z = cam[2]*x + cam[5]*y + cam[8]*z + cam[11];

		if(panoramic){
			//panoramic camera model
			y = (y/sqrt(z*z + x*x));
			x = atan2(x,z);

		}
		else{
			//pin point camera model
			y = y/z;
			x = x/z;
		}
	}

	//output points
	pointsOut[i + 0*numPoints] = x;
	pointsOut[i + 1*numPoints] = y;
}