#include <mex.h>

#include "Tforms.h"
#include "ScanList.h"
#include "ImageList.h"
#include "Kernels.h"

void Tforms::addTforms(thrust::device_vector<float> tformDIn, size_t tformSizeX, size_t tformSizeY){
	if(tformDIn.size() != (tformSizeX*tformSizeY)){
		mexErrMsgTxt("Error input tform matricies must be same size as given dimensions in size");
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = tformSizeX;
	tformD.back().tformSizeY = tformSizeY;
}

void Tforms::addTforms(thrust::host_vector<float> tformDIn, size_t tformSizeX, size_t tformSizeY){
	if(tformDIn.size() != (tformSizeX*tformSizeY)){
		mexErrMsgTxt("Error input tform matricies must be same size as given dimensions in size");
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = tformSizeX;
	tformD.back().tformSizeY = tformSizeY;
}

void Tforms::removeAllTforms(void){
	tformD.clear();
}

float* Tforms::getTformP(size_t idx){
	if(tformD.size() <= idx){
		std::ostringstream err;
		err << "Cannot get pointer to element " << idx << " as only " << tformD.size() << " elements exist";
		mexErrMsgTxt(err.str().c_str());
		return NULL;
	}
	return thrust::raw_pointer_cast(&(tformD[idx].tform[0]));
}

size_t Tforms::getTformSize(size_t idx){
	if(tformD.size() <= idx){
		std::ostringstream err; err << "Cannot get element " << idx << " as only " << tformD.size() << " elements exist";
		mexErrMsgTxt(err.str().c_str());
		return 0;
	}
	return (tformD[idx].tformSizeX * tformD[idx].tformSizeY);
}

void Tforms::transform(ScanList scansIn, std::vector<float*>& locOut, Cameras cam, size_t tformIdx, size_t camIdx, size_t scanIdx, hipStream_t stream){};

void CameraTforms::addTforms(thrust::device_vector<float> tformDIn){
	if(tformDIn.size() != 16){
		std::ostringstream err; err << "Error input tform matricies must be same size as given dimensions in size";
		mexErrMsgTxt(err.str().c_str());
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 4;
	tformD.back().tformSizeY = 4;
}

void CameraTforms::addTforms(thrust::host_vector<float> tformDIn){
	if(tformDIn.size() != 16){
		std::ostringstream err; err << "Error input tform matricies must be same size as given dimensions in size";
		mexErrMsgTxt(err.str().c_str());
		return;
	}
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 4;
	tformD.back().tformSizeY = 4;
}

void CameraTforms::transform(ScanList scansIn, std::vector<float*>& locOut, Cameras cam, size_t tformIdx, size_t camIdx, size_t scanIdx, hipStream_t stream){

	CameraTransformKernel<<<gridSize(scansIn.getNumPoints(scanIdx)), BLOCK_SIZE, 0, stream>>>(
		getTformP(tformIdx),
		cam.getCamP(camIdx),
		cam.getPanoramic(camIdx),
		scansIn.getLP(scanIdx,0),
		scansIn.getLP(scanIdx,1),
		scansIn.getLP(scanIdx,2),
		scansIn.getNumPoints(scanIdx),
		locOut[0],
		locOut[1]);

	CudaCheckError();
}

void AffineTforms::addTforms(thrust::host_vector<float> tformDIn){
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 3;
	tformD.back().tformSizeY = 3;
}

void AffineTforms::addTforms(thrust::device_vector<float> tformDIn){
	tform tformIn;
	tformD.push_back(tformIn);
	tformD.back().tform = tformDIn;
	tformD.back().tformSizeX = 3;
	tformD.back().tformSizeY = 3;
}

void AffineTforms::transform(ScanList scansIn, std::vector<float*>& locOut, Cameras cam, size_t tformIdx, size_t camIdx, size_t scanIdx, hipStream_t stream){
	AffineTransformKernel<<<gridSize(scansIn.getNumPoints(scanIdx)), BLOCK_SIZE, 0, stream>>>(
		getTformP(tformIdx),
		scansIn.getLP(scanIdx,0),
		scansIn.getLP(scanIdx,1),
		scansIn.getNumPoints(scanIdx),
		locOut[0],
		locOut[1]);

	CudaCheckError();
}
